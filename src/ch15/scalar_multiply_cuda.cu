/*
 * Copyright (c) 2020, Dive into Systems, LLC
 * https://diveintosystems.org/
 *
 * Example CUDA program that does scalar vector multiply
 *
 * to compile: 
 *  nvcc -g -G -o scalar_multiply_cuda  scalar_multiply_cuda.cu -lstdc++
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

/* define these so that N is evenly divided by BLOCK_SIZE */
#define BLOCK_SIZE       64    /* threads per block */ 
#define N              10240   /* vector size */ 
#define DEFAULT_VALUE  3       /* default scalar value */

/******** prototypes ************/
/* host-side: vector initialization function */
void init_array(int *vector, int size, int step);
/* host-size: print out some vector values */
void print_some(int *vector, int size, int step);
/* cuda scalar multiply kernel */
__global__ void scalar_multiply(int *array, int scalar); 


/***************************************************/
// host-side main:
int main(int argc, char **argv) {

  int *vector, *dev_vector, scalar;

  scalar = DEFAULT_VALUE;
  if(argc == 2) {
    scalar = atoi(argv[1]);
  }

  // allocate host memory space for the vector
  vector = (int *)malloc(sizeof(int)*N);
  if (vector == NULL) {
    printf("Error: malloc failed\n");
    exit(1);
  }
  // initialize vector in host memory
  init_array(vector, N, 7);

  // allocate GPU device memory for vector
  if (hipMalloc(&dev_vector, sizeof(int)*N) != hipSuccess) {
    printf("Error: hipMalloc failed\n");
    exit(1);
  }

  // copy host vector to device memory
  if (hipMemcpy(dev_vector, vector, sizeof(int)*N, hipMemcpyHostToDevice)
      != hipSuccess) 
  {
    printf("Error: hipMemcpy host to dev failed\n");
    exit(1);
  }

  // call the CUDA scalar_multiply kernel
  // specify the 1D blocks/grid and 1D threads/block layout in the call
  scalar_multiply<<<(N/BLOCK_SIZE), BLOCK_SIZE>>>(dev_vector, scalar);

  // copy device vector to host memory
  if (hipMemcpy(vector, dev_vector, sizeof(int)*N, hipMemcpyDeviceToHost)
      != hipSuccess) 
  {
    printf("Error: hipMemcpy dev to host failed\n");
    exit(1);
  }

  // print out some of the result to check
  print_some(vector, N, 117);

  // free space
  hipFree(dev_vector);
  free(vector);

  return 0;
}
/***************************************************/
/*
 * CUDA kernel function that performs scalar multiply 
 * of a vector on the GPU device
 *
 * This assumes that there are enough threads to associated
 * each array[i] element with a signal thread
 * (in general, each thread would be responsible for a set of data elements)
 */
__global__ void scalar_multiply(int *array, int scalar) {

  int index;

  // get the thread's index value based on its 
  // position in the enclosing block and grid
  index = blockIdx.x * blockDim.x + threadIdx.x;

  array[index] = array[index] * scalar;
}
/***************************************************/
/* host-side: vector initialization function */
void init_array(int *vector, int size, int step) {

  int i;

  for(i=0; i < size; i++) { 
    vector[i] = i % step; 
  }

}
/* host-size: print out some vector values */
void print_some(int *vector, int size, int step) {

  int i;

  i = 0;
  while( i < size) {
    printf("v[%d] = %d\n", i, vector[i]);
    i = i + step;
  }

}



